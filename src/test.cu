#include "hip/hip_runtime.h"
#include<stdio.h>
#include<H5Cpp.h>

#define KEY_HYPOTHESIS 256
#define NUM_TRACES 1000
#define BLOCKSIZE 32

typedef struct ascad_metadata {
    unsigned char plaintext[16];
    unsigned char ciphertext[16];
    unsigned char key[16];
    unsigned char masks[16];
    unsigned int  desync;
} ascad_metadata;

__global__ void create_model(uint8_t *d_model, uint8_t *d_plaintexts){
    uint8_t Sbox[] = {  
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
        };
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * KEY_HYPOTHESIS + ix;             //何スレッド目か

    if(ix < KEY_HYPOTHESIS && iy < NUM_TRACES)
        d_model[idx] = HW(Sbox[d_plaintexts[iy] ^ ix]);
}

__device__ uint8_t HW(uint8_t x)
{
    x = x - ((x >> 1) & 0x55555555);
    x = (x & 0x33333333) + ((x << 2) & 0x33333333);

    return ((x + (x >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
}

__global__ void transpose_model(uint8_t *out, uint8_t *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < nx && iy << ny){
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

__global__ void correlation(float *d_corr, int8_t *d_trases_t, uint8_t *d_model_t, int T, int D){
    // T is n_pois, D is n_traces? 
    // listing 4.4: Computation of the correlation matrix for a first order CPA attack based on [CDOR09].
    __shared__ float Xs[BLOCKSIZE][BLOCKSIZE];
    __shared__ float Ys[BLOCKSIZE][BLOCKSIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int xBegin = bx * BLOCKSIZE * D;
    int yBegin = by * BLOCKSIZE * D;
    int yEnd = yBegin + D - 1;

    int x, y, k, o;
    float a1, a2, a3, a4, a5;
    float avgX, avgY, varX, varY, cov, rho;

    a1 = a2 = a3 = a4 = a5 = 0.0;

    for(y = yBegin, x = xBegin; y <= yEnd; y += BLOCKSIZE, x += BLOCKSIZE){
        Xs[tx][ty] = d_model_t[x + ty * D + tx];
        Ys[ty][tx] = d_trases_t[y + ty * D + tx];

        __syncthreads();

        for(k = 0; k < BLOCKSIZE; k++){
            a1 += Xs[k][tx];
            a2 += Ys[ty][k];
            a3 += Xs[k][tx] * Xs[k][tx];
            a4 += Ys[ty][k] * Ys[ty][k];
            a5 += Xs[k][tx] * Ys[ty][k];
        }

        __syncthreads();
    }
    
    avgX = a1 / D;
    avgY = a2 / D;

    varX = (a3 - avgX * avgX * D) / (D - 1);
    varY = (a4 - avgY * avgY * D) / (D - 1);
    cov = (a5 - avgX * avgY * D) / (D - 1);

    rho - cov / sqrtf(varX * varY);
    o = bx * BLOCKSIZE * T + tx * T + by * BLOCKSIZE + ty;

    d_corr[o] = rho;
}

__global__ void merge_sums(float *d_centr_sum_t_q1, float *d_centr_sum_1_q1, float *d_mean_t_q1, float *d_mean_1_q1,
                            float *d_centr_sum_t_q2, float *d_centr_sum_1_q2, float *d_mean_t_q2, float *d_mean_1_q2,
                            int D, int T, int K, int iteration){
                                
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;

    float mean_t_q1, mean_1_q1, centr_sum_t_q1, centr_sum_1_q1;
    float mean_t_q2, mean_1_q2, centr_sum_t_q2, centr_sum_1_q2;
    float mean_t_q, mean_1_q, centr_sum_t_q, centr_sum_1_q;
    float delta_1, delta_t, delta_n_1, delta_n_t;

    int n1, n2, n;

    n1 = D * iteration;
    n2 = D;
    n = n1 + n2;

    if(tidx < K){
        centr_sum_1_q1 = d_centr_sum_1_q1[tidx];
        centr_sum_1_q2 = d_centr_sum_1_q2[tidx];

        mean_1_q1 = d_mean_1_q1[tidx];
        mean_1_q2 = d_mean_1_q2[tidx];

        delta_1 = mean_1_q2 - mean_1_q1;
        delta_n_1 = delta_1 / n;

        centr_sum_1_q = centr_sum_1_q1 + centr_sum_1_q2 + n1 * n2 * delta_1 * delta_n_1;
        d_centr_sum_1_q1[tidx] = centr_sum_1_q;

        mean_1_q = mean_1_q1 + n2 * delta_n_1;
        d_mean_1_q1[tidx] = mean_1_q;
    }

    if(tidx < T){
        centr_sum_t_q1 = d_centr_sum_t_q1[tidx];
        centr_sum_t_q2 = d_centr_sum_t_q2[tidx];

        mean_t_q1 = d_mean_t_q1[tidx];
        mean_t_q2 = d_mean_t_q2[tidx];

        delta_t = mean_t_q2 - mean_t_q1;
        delta_n_t = delta_t / n;

        centr_sum_t_q = centr_sum_t_q1 + centr_sum_t_q2 + n1 * n2 * delta_t * delta_n_t;
        d_centr_sum_t_q1[tidx] = centr_sum_t_q;

        mean_t_q = mean_t_q1 + n2 * delta_n_t;
        d_mean_t_q1[tidx] = mean_t_q;
    }
}

__global__ void merge_adj_sum(float *d_adj_centr_sum_q1, float *d_mean_l_q1, float *d_mean_t_q1, 
                            float *d_adj_centr_sum_q2, float *d_mean_l_q2, float *d_mean_t_q2,
                            int D, int T, int K, int iteration)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;

    float mean_t_q1, mean_l_q1, adj_centr_sum_q1;
    float mean_t_q2, mean_l_q2, adj_centr_sum_q2;
    float adj_centr_sum_q;
    float delta_t, delta_l;

    int n1, n2, n, index;

    if(tidx < T){
        n1 = D * iteration;
        n2 = D;
        n = n1 + n2;

        for(int k = 0; k < K; k++){
            index = k * T + tidx;

            mean_t_q1 = d_mean_t_q1[tidx];
            mean_t_q2 = d_mean_t_q2[tidx];

            mean_l_q1 = d_mean_l_q1[k];
            mean_l_q2 = d_mean_l_q2[k];

            adj_centr_sum_q1 = d_adj_centr_sum_q1[index];
            adj_centr_sum_q2 = d_adj_centr_sum_q2[index];

            delta_l = mean_l_q2 - mean_l_q1;
            delta_t = mean_t_q2 - mean_t_q1;

            adj_centr_sum_q = adj_centr_sum_q1 + adj_centr_sum_q2 + ((n1 * n2) / n) * delta_t * delta_l;
            d_adj_centr_sum_q1[index] = adj_centr_sum_q;
        }
    }
}


int main(int argc, char **argv){

	H5::H5File file("./my_ASCAD.h5", H5F_ACC_RDONLY);
    H5::Group group = file.openGroup("Attack_traces"); 
	H5::DataSet dataset = group.openDataSet("traces"); 
	H5::DataSpace dataspace = dataset.getSpace();
    int n_traces, n_pois;
    file.close();

    hsize_t dims[dataspace.getSimpleExtentNdims()];
    dataspace.getSimpleExtentDims( dims );
    n_traces = dims[0];
    n_pois = dims[1];

    char *traces = new char [n_traces * n_pois];
    printf("Traces dimension: %llu x %llu\n", dims[0], dims[1]);
    dataset.read( traces, dataset.getDataType() );

	dataset = group.openDataSet("metadata"); 
    ascad_metadata *metadata = new ascad_metadata [n_traces];
    unsigned char *plaintexts = new unsigned char[n_traces*16];

    dataset.read(metadata, dataset.getDataType() );
    for(int i=0; i<n_traces; i++){
        memcpy(plaintexts + 16*i, metadata[i].plaintext, 16);
    }

    for(int i = 0; i < 8; i++){
        for(int j = 0; j < 16; j++){
            printf("%d ", *(plaintexts + 16*i + j));
        }
        printf("\n");
    }

    // helloFromGPU<<<1, 10>>>();
    // hipDeviceReset();
    delete traces, metadata;
    dataset.close();
    return 0;
}