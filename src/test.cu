#include "hip/hip_runtime.h"
#include <stdio.h>
#include<H5Cpp.h>
#include<hdf5.h>

__global__ void helloFromGPU(){
    printf("Hello World rom GPU!\n");
}

int main(int argc, char **argv){
    printf("Hello World from CPU!\n");

	H5::H5File file("../ASCAD-master/ATMEGA_AES_v1/ATM_AES_v1_fixed_key/ASCAD_data/ASCAD_databases/my_ASCAD.h5", H5F_ACC_RDONLY);
    H5::Group group= file.openGroup("Attack_traces"); 
	H5::DataSet dataset = group.openDataSet("traces"); 
	H5::DataSpace dataspace = dataset.getSpace();

    int n_traces = dataspace.getSimpleExtentNdims();

    printf("%d\n", n_traces);

    // helloFromGPU<<<1, 10>>>();
    // hipDeviceReset();
    return 0;
}