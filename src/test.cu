#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hdf5.h"

__global__ void helloFromGPU(){
    printf("Hello World rom GPU!\n");
}

int main(int argc, char **argv){
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}